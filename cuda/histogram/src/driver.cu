#include <iostream>
#include <hip/hip_runtime.h>
#include "../include/kernel.cuh"

const int IMG_SIZE = 1024 * 1024;
const int HIST_SIZE = 256;

int main() {
    unsigned char* h_image = new unsigned char[IMG_SIZE];
    for (int i = 0; i < IMG_SIZE; i++)
        h_image[i] = rand() % 256;

    unsigned char* d_image;
    int* d_histogram;
    hipMalloc(&d_image, IMG_SIZE);
    hipMalloc(&d_histogram, HIST_SIZE * sizeof(int));
    hipMemcpy(d_image, h_image, IMG_SIZE, hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, HIST_SIZE * sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (IMG_SIZE + threadsPerBlock - 1) / threadsPerBlock;
    compute_histogram<<<blocks, threadsPerBlock>>>(d_image, d_histogram, IMG_SIZE);
    hipDeviceSynchronize();

    int h_histogram[HIST_SIZE];
    hipMemcpy(h_histogram, d_histogram, HIST_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i)
        std::cout << "Bin " << i << ": " << h_histogram[i] << std::endl;

    delete[] h_image;
    hipFree(d_image);
    hipFree(d_histogram);
    return 0;
}

