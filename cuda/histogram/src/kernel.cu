#include "hip/hip_runtime.h"
#include "../include/kernel.cuh"

__global__ void compute_histogram(const unsigned char* image, int* histogram, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        unsigned char pixel = image[idx];
        atomicAdd(&histogram[pixel], 1);
    }
}

